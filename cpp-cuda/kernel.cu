#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <cmath>

__global__ void matmul_kernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}


hipError_t matmul_gpu(const float* lhs, const float* rhs, float* out, unsigned int n);


int main()
{   
    printf("Started\n");
    const int n = 1024*1024*16*16;
    float *a = (float*)malloc(n * sizeof(float));
    float *b = (float*)malloc(n * sizeof(float));
    srand(42);
    #pragma omp parallel for
    for (size_t i = 0; i != n; ++i) {
        a[i] = (rand() % 10) - 10;
        b[i] = (rand() % 10) - 10;
    }
    float *c = (float*)malloc(n * sizeof(float));
    printf("Init finalized\n");
    clock_t begin = clock();
    hipError_t cudaStatus = matmul_gpu(a, b, c, n);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("c[0] = %.1f\n", c[0]);
    printf("time spent = %.6fs\n", time_spent);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    printf("Press Any Key to Exit\n");
    getchar();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t matmul_gpu(const float *lhs, const float *rhs, float *out, unsigned int n)
{
    float *lhs_gpu = 0;
    float *rhs_gpu = 0;
    float *out_gpu = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&out_gpu, n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&lhs_gpu, n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&rhs_gpu, n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(lhs_gpu, lhs, n * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(rhs_gpu, rhs, n * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    int block_size;
    int grid_size;
    int min_grid_size;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, matmul_kernel, 0, 0);
    grid_size = (n + block_size - 1) / block_size;
    printf("using %d blocks and %d threads per block\n", grid_size, block_size);

    clock_t begin = clock();
    matmul_kernel<<<grid_size, block_size>>>(lhs_gpu, rhs_gpu, out_gpu, static_cast<int>(sqrt(n)));
    hipDeviceSynchronize();
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("time spent on kernel = %.6fs\n", time_spent);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(out, out_gpu, n * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(out_gpu);
    hipFree(lhs_gpu);
    hipFree(rhs_gpu);
    
    return cudaStatus;
}
